#include "hip/hip_runtime.h"
#include "funcionesD.h"

void GAUSS(EntradaCuerpo* entradaCuerpo, double punt[7][3])
{
    //* Declaracion de variables
    // Aprovechamos que son punteros para direccionarlos a las matrices del cuerpo.
    // Nos ahorra muchas modificaciones.
    double** AE_T = entradaCuerpo->AE_T;
    double** BE_T = entradaCuerpo->BE_T;
    double** AT_T = entradaCuerpo->AT_T;
    double** BT_T = entradaCuerpo->BT_T;
    double** CTE_T = entradaCuerpo->CTE_T;
    double** DTE_T = entradaCuerpo->DTE_T;

    double ** AE = entradaCuerpo->AE;
    double ** BE = entradaCuerpo->BE;
    double &AT = entradaCuerpo->AT;
    double &BT = entradaCuerpo->BT;
    double * CTE = entradaCuerpo->CTE;
    double * DTTE = entradaCuerpo->DTTE;
    double ** DTE = entradaCuerpo->DTE;


    // Reemplazamos las referencias globales de variables.h por las del cuerpo. No son punteros, pero el '&' nos permite
    // vincular variables en C++:
    int &nelT = entradaCuerpo->nelT;
    double &GT = entradaCuerpo->GT;
    double &nuT = entradaCuerpo->nuT;
    double &alT = entradaCuerpo->alT;

    double ** exT = entradaCuerpo->exT;
    int    ** conT = entradaCuerpo->conT;
    double ** ndT = entradaCuerpo->ndT;
    double ** locT = entradaCuerpo->locT;

    double * ndCol = entradaCuerpo->ndCol;
    double ** extr = entradaCuerpo->extr;

    int &tipoSimetria = entradaCuerpo->tipoSimetria;
    char &intenum = entradaCuerpo->intenum;

    double  cnat[7][3]; //valores de las coordenadas naturales
    double  acnat;  //auxiliares
    double  bcnat;  //auxiliares
    int  i,p,j;

    //* Coordenadas naturales
    acnat=(6.0+sqrt(15.0))/21.0;
    bcnat=4.0/7.0-acnat;

    cnat[0][0]=1.0/3.0;
    cnat[1][0]=1.0-2.0*acnat;
    cnat[2][0]=acnat;
    cnat[3][0]=acnat;
    cnat[4][0]=1.0-2.0*bcnat;
    cnat[5][0]=bcnat;
    cnat[6][0]=bcnat;

    cnat[0][1]=1.0/3.0;
    cnat[1][1]=acnat;
    cnat[2][1]=1.0-2.0*acnat;
    cnat[3][1]=acnat;
    cnat[4][1]=bcnat;
    cnat[5][1]=1.0-2.0*bcnat;
    cnat[6][1]=bcnat;

    cnat[0][2]=1.0/3.0;
    cnat[1][2]=acnat;
    cnat[2][2]=acnat;
    cnat[3][2]=1.0-2.0*acnat;
    cnat[4][2]=bcnat;
    cnat[5][2]=bcnat;
    cnat[6][2]=1.0-2.0*bcnat;

    //* Inicializacion de los puntos de integracion
    for( p=0; p<7; p++)        
        for( i=0; i<3; i++)            
            punt[p][i]=0.0;

    //* Calculo de los puntos de integracion
    for( p=0; p<7; p++)        
        for( i=0; i<3; i++)            
            for( j=0; j<3; j++)                
                punt[p][i]=punt[p][i]+cnat[p][j]*extr[j][i];

    return;
}