#include "hip/hip_runtime.h"
#include "funcionesMain.h"

void CODIFICADA()
{
    //* Declaracion de variables
    
    
    int  bContacto; //Indica si es superficie de contacto
    int  nele,nex,nexpc,nin;    //Número de elementos y extremos actuales. Número de extremos en la zona de contacto
    int  ifla1; //Flag para continuar o no la lectura
    int  tipoS; //Tipo de superficie
    int  i,j;   //Auxiliares
    //* Lee los datos relativos a la zona de contacto
    nele=1;
    nex=1;
    ifla1=0;
    bContacto=1;
    while(ifla1 == 0)
    {
        leeEntero(in1,&tipoS);leeEntero(in1,&ifla1);leeLinea(in1);
        //* Segun el tipo de contorno sigue un camino u otro
        if(tipoS == 0)        
        {
            //* Contacto S4-S4
            SUPERFICIE_CUATRO(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else if(tipoS == 1)        
        {
            //* Contacto S3-S3
            SUPERFICIE_TRES(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else        
        {
            //* Contacto entre otros tipos de superficies
            printf(" **ERROR**: SUPERFICIE DE CONTACTO NO PROGRAMADA\n");
            printf("REVISE LA ENTRADA DE DATOS\n"); enExcepcion=1;return;
        }
    }
    //* Se ha terminado la zona de contacto
    nelpc=nele-1;
    nexpc=nex-1;
    //* Lee codigos de la zona de contacto
    nin=1;
    LEE_CODIGOS(&nin,&nelpc);if(enExcepcion==1)return;
    //* Inicializa condiciones de contorno de la zona de contacto
    for( i=1; i<=nelpc; i++)      
    {
        codA[i-1][1-1]=codB[i-1][1-1];
        codA[i-1][2-1]=codB[i-1][2-1];
        for( j=1; j<=10; j++)        
        {
            ccA[i-1][j-1]=0.;
            ccB[i-1][j-1]=0.;
        }
    }
    //* Lee los datos relativos a la zona libre de "A"
    ifla1=0;
    bContacto=0;
    while(ifla1 == 0)
    {
        leeEntero(in1,&tipoS);leeEntero(in1,&ifla1);leeLinea(in1);
        //* Segun el tipo de contorno sigue un camino u otro
        if(tipoS == 0)        
        {
            //* Superficie S4
            SUPERFICIE_CUATRO(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else if(tipoS == 1)        
        {
            //* Superficie S3-S3
            SUPERFICIE_TRES(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else        
        {
            //* Otros tipos de superficies
            printf(" **ERROR**: SUPERFICIE NO PROGRAMADA EN SOLIDO A\n");
            printf("REVISE LA ENTRADA DE DATOS\n"); enExcepcion=1;return;
        }
    }
    //* Se ha terminado la zona libre de "A"
    nelA=nele-1;
    nexA=nex-1;
    //* Lee codigos de la zona libre de A
    nin=nelpc+1;
    LEE_CODIGOS(&nin,&nelA);if(enExcepcion==1)return;
    //* Lee condiciones de contorno de la zona libre de A
    LEE_CC(&nin,&nelA);if(enExcepcion==1)return;
    //* Asigna valores a variables relativas a "A"
    for( i=nexpc+1; i<=nexA; i++)      
    {
        for( j=1; j<=3; j++)        
        {
            exA[i-1][j-1]=exB[i-1][j-1];
        }
    }
    for( i=nelpc+1; i<=nelA; i++)      
    {
        codA[i-1][1-1]=codB[i-1][1-1];
        codA[i-1][2-1]=codB[i-1][2-1];
        for( j=1; j<=3; j++)        
        {
            conA[i-1][j-1]=conB[i-1][j-1];
        }
        for( j=1; j<=10; j++)        
        {
            ccA[i-1][j-1]=ccB[i-1][j-1];
        }

        for( j=1; j<=9; j++)        
        {
            locA[i-1][j-1]=locB[i-1][j-1];
        }
    }
    
    
#ifdef DEBUG
    printf("[funcionesMain.c] locA address: %p\n", locA);
    printf("[funcionesMain.c] locA[0][0] value: %f\n", locA[0][0]);
#endif
    
    //*    WRITE(*,*) 'SOLIDO B'
    //* Lee los datos relativos a la zona libre de "B"
    nele=nelpc+1;
    nex=nexpc+1;
    ifla1=0;
    while(ifla1 == 0)
    {
        leeEntero(in1,&tipoS);leeEntero(in1,&ifla1);leeLinea(in1);
        //* Segun el tipo de contorno sigue un camino u otro
        if(tipoS == 0)        
        {
            //* Superficie S4
            SUPERFICIE_CUATRO(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else if(tipoS == 1)        
        {
            //* Superficie S3-S3
            SUPERFICIE_TRES(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else        
        {
            //* Otros tipos de superficies
            printf(" **ERROR**: SUPERFICIE NO PROGRAMADA EN SOLIDO A\n");
            printf("REVISE LA ENTRADA DE DATOS\n"); enExcepcion=1;return;
        }
    }
    //* Se ha terminado la zona libre de "B"
    nelB=nele-1;
    nexB=nex-1;
    //* Lee codigos de la zona libre de B
    nin=nelpc+1;
    LEE_CODIGOS(&nin,&nelB);if(enExcepcion==1)return;
    //* Lee condiciones de contorno de la zona libre de B
    LEE_CC(&nin,&nelB);if(enExcepcion==1)return;
    //* Calcula los nodos de ambos cuerpos
    CALCNODOS();if(enExcepcion==1)return;
    //* Calcula el sistema local de coordenadas de ambos cuerpos
    SISTLOCAL();if(enExcepcion==1)return;
    return;
}



void SEMICODIFICADA()
{
    //* Declaracion de variables
    
    
    
    
    //*       INCOMPLETA
    
    
    return;
}



void NODOS_CARGADOS()
{
    //* Declaracion de variables
    
    
    int  nd;    //Auxiliares
    int  i;
    //* Para el solido A
    nelwA=0;
    for( nd=nelpc+1; nd<=nelA; nd++)      
    {
        for( i=1; i<=10; i++)       
        {
            if(ccA[nd-1][i-1] != 0.0 )         
            {
                nelwA=nelwA+1;
                break;
            }
        }
    }
    //* Para el solido B
    nelwB=0;
    for( nd=nelpc+1; nd<=nelB; nd++)      
    {
        for( i=1; i<=10; i++)       
        {
            if(ccB[nd-1][i-1] != 0.0 )         
            {
                nelwB=nelwB+1;
                break;
            }
        }
    }
    return;
}



void DIR_CONTACTO()
{
    //* Declaracion de variables
    
    
    int  nd;    //Auxiliares
    int  i;
    double  modulo;
    //* Para todos los elementos de la zona potencial de contacto
    for( nd=1; nd<=nelpc; nd++)      
    {
        modulo=0.0;
        for( i=1; i<=3; i++)        
        {
            modulo=pow(modulo+(locA[nd-1][i-1]-locB[nd-1][i-1]),2);
        }
        modulo=sqrt(modulo);
        if(modulo != 0.0 )        
        {
            locA[nd-1][1-1]=(locA[nd-1][1-1]-locB[nd-1][1-1])/modulo;
            locB[nd-1][1-1]=-locA[nd-1][1-1];
            locA[nd-1][2-1]=(locA[nd-1][2-1]-locB[nd-1][2-1])/modulo;
            locB[nd-1][2-1]=-locA[nd-1][2-1];
            locA[nd-1][3-1]=(locA[nd-1][3-1]-locB[nd-1][3-1])/modulo;
            locB[nd-1][3-1]=-locA[nd-1][3-1];
        }
    }
    return;
}



void GAP_INICIAL()
{
    //* Declaracion de variables
    
    int  nd;    //Auxiliares
    double  modulo;
    //* Para todos los elementos de la zona potencial de contacto
    for( nd=1; nd<=nelpc; nd++)      
    {
        //* Calcula el modulo para ver si es nulo
        modulo=sqrt(pow((ndA[nd-1][1-1]-ndB[nd-1][1-1]),2)+pow((ndA[nd-1][2-1]-ndB[nd-1][2-1]),2)+pow((ndA[nd-1][3-1]-ndB[nd-1][3-1]),2));
        if(modulo == 0. )        
        {
            //* Separacion nula
            gap[nd-1]=0.;
        }
        else        
        {
            //* Calcula la separacion sobre la direccion de contacto
            gap[nd-1]=(ndA[nd-1][1-1]-ndB[nd-1][1-1])*locA[nd-1][1-1]+(ndA[nd-1][2-1]-ndB[nd-1][2-1])*locA[nd-1][2-1]+(ndA[nd-1][3-1]-ndB[nd-1][3-1])*locA[nd-1][3-1];
        }
    }
    return;
}




void INTEGRAL()
{
    //* Declaracion de variables
    
    
    int  el,ex; //Auxiliares
    int  j;
    float tiniA,tiniB,tfinA,tfinB;
    
    out11=81;
    out13=83;
    out14=84;
    
    printf("==================== Entrando en INTEGRAL ====================\n");
    //* Lee los datos del problema

    entradaCuerpoA.nexT = nexA;
    entradaCuerpoA.nelT = nelA;
    entradaCuerpoA.ET = EA;
    entradaCuerpoA.alT = alA;
    entradaCuerpoA.nuT = nuA;
    entradaCuerpoA.GT = EA/(2.0*(1.0+nuA));
    EntradaCuerpo entradaCuerpoACuda = entradaCuerpoA;

    inicializar(&entradaCuerpoA, HOST);
    inicializar(&entradaCuerpoACuda, CUDA);

    copiarMatriz((void**)entradaCuerpoA.exT, (void**)exA, sizeof(double), nexA, 3, MEMCPY_HOST_TO_HOST);
    copiarMatriz((void**)entradaCuerpoA.conT, (void**)conA, sizeof(double), nelA, 3, MEMCPY_HOST_TO_HOST);
    copiarMatriz((void**)entradaCuerpoA.ndT, (void**)ndA, sizeof(double), nelA, 3, MEMCPY_HOST_TO_HOST);
    copiarMatriz((void**)entradaCuerpoA.locT, (void**)locA, sizeof(double), nelA, 9, MEMCPY_HOST_TO_HOST);

    copiarMatriz((void**)entradaCuerpoACuda.exT, (void**)exA, sizeof(double), nexA, 3, hipMemcpyHostToDevice);
    copiarMatriz((void**)entradaCuerpoACuda.conT, (void**)conA, sizeof(double), nelA, 3, hipMemcpyHostToDevice);
    copiarMatriz((void**)entradaCuerpoACuda.ndT, (void**)ndA, sizeof(double), nelA, 3, hipMemcpyHostToDevice);
    copiarMatriz((void**)entradaCuerpoACuda.locT, (void**)locA, sizeof(double), nelA, 9, hipMemcpyHostToDevice);




    //* Abre ficheros
    
    //* Calcula coeficientes del cuerpo "A"
    printf("++++++++ Calcula coeficientes del cuerpo \"A\" ++++++++\n");
    tiniA = clock()/CLOCKS_PER_SEC;
    printf("[%.3fs] Inicio calculo coeficientes.\n",tiniA);

    COEFICIENTES(AE_A,BE_A,AT_A,BT_A,CTE_A,DTE_A);if(enExcepcion==1)return;
    
    tfinA = clock()/CLOCKS_PER_SEC;
    printf("[%.3fs] Final calculo coeficientes.\n",tfinA);
    printf("\t** Tiempo Total= %f segundos\n",tfinA-tiniA);



    /*entradaCuerpoB.nexT = nexB;
    entradaCuerpoB.nelT = nelB;
    entradaCuerpoB.ET = EB;
    entradaCuerpoB.alT = alB;
    entradaCuerpoB.nuT = nuB;
    entradaCuerpoB.GT = EB/(2.0*(1.0+nuB));

    inicializar(&entradaCuerpoB, CUDA);
    copiarMatriz(entradaCuerpoB.exT, exB, sizeof(double), nexB, 3, hipMemcpyHostToDevice);
    copiarMatriz(entradaCuerpoB.conT, conB, sizeof(double), nelB, 3, hipMemcpyHostToDevice);
    copiarMatriz(entradaCuerpoB.ndT, ndB, sizeof(double), nelB, 3, hipMemcpyHostToDevice);
    copiarMatriz(entradaCuerpoB.locT, locB, sizeof(double), locB, 9, hipMemcpyHostToDevice);

    //* Calcula coeficientes del cuerpo "B"
    printf("+++++++ Calcula coeficientes del cuerpo \"B\" +++++++\n");
    tiniB = clock()/CLOCKS_PER_SEC;
    printf("[%.3fs] Inicio calculo coeficientes.\n",tiniB);

    COEFICIENTES(AE_B,BE_B,AT_B,BT_B,CTE_B,DTE_B);if(enExcepcion==1)return;


    //* Final del programa
    
    tfinB = clock()/CLOCKS_PER_SEC;
    printf("[%.3fs] Final calculo coeficientes.\n",tfinB);
    printf("\t** Tiempo Total= %f segundos\n",tfinB-tiniB);
    
    printf("==================== Saliendo de INTEGRAL ===================\n");
    printf("TIEMPOOOOOOOOO: %fs\n",(tfinA-tiniA)+(tfinB-tiniB));*/
}


void TERMICO ()
{
    //* Declaracion de variables    
    //* Inicializa variables
    
    
    //* Comienzo del programa
    printf(" ==================== Entrando en TERMICO ====================\n");
    
    //* Se llama a la subrutina que trae los coeficientes   y monta el sistema
    MONTAJETER();if(enExcepcion==1)return;
    
    //* resuelve el sistema por el metodo de GAUSS
    GAUSS_SOLU();if(enExcepcion==1)return;
    //* Interpretacion de resultados (condiciones de contorno y contacto)
    CONTORNOTER();if(enExcepcion==1)return;
    //* Salida de resultados a ficheros
    SALIDATER();if(enExcepcion==1)return;
    //* Final del programa
    printf(" ==================== Saliendo de TERMICO ====================\n");
    return;
}



void MONTAJE()
{
    //* Declaracion de variables
    
    int i,j,nd,el,m;
    //**** NOTA : La matriz se va construyendo por filas.
    
    //* Inicializa el vector de cargas y la matriz de coeficientes
    for( i=1; i<=ngl; i++)      
    {
        b[i-1]=0.0;
        for( j=1; j<=ngl; j++)            
        {
            a[i-1][j-1]=0.0;
        }
    }
    //* Abre ficheros de coeficientes elásticos del cuerpo A
    
    //* Monta la parte correspondiente al cuerpo A del problema elastico
    printf("  Monta la parte correspondiente al cuerpo A del problema elastico\n");
    for( nd=1; nd<=nelA; nd++)      
    {
        for( el=1; el<=nelA; el++)            
        {
            //* Lee los coeficientes
            
            int arTemp_0;int arTemp_1;int arTemp_2;int arTemp_3;arTemp_2 = el*3 -2-1; for(arTemp_0=1;arTemp_0<=3;arTemp_0+=1)
            {
                arTemp_2 += 1;arTemp_3 = nd*3 -2-1; for(arTemp_1=1;arTemp_1<=3;arTemp_1+=1)
                {
                    arTemp_3 += 1;AE[arTemp_0-1][arTemp_1-1] =  AE_A[arTemp_2-1][arTemp_3-1];
                }
            }
            ;
            int arTemp_4;int arTemp_5;int arTemp_6;int arTemp_7;arTemp_6 = el*3 -2-1; for(arTemp_4=1;arTemp_4<=3;arTemp_4+=1)
            {
                arTemp_6 += 1;arTemp_7 = nd*3 -2-1; for(arTemp_5=1;arTemp_5<=3;arTemp_5+=1)
                {
                    arTemp_7 += 1;BE[arTemp_4-1][arTemp_5-1] =  BE_A[arTemp_6-1][arTemp_7-1];
                }
            }
            ;
            
            //* Mira si el nodo el pertenace a la zona actual de contacto
            if(codA[el-1][1-1] < 9)                  
            {
                //* No pertenece
                cod=codA[el-1][1-1];
                nel=0.0;
                for( m=1; m<=6; m++)                        
                {
                    cc[m-1]=ccA[el-1][m-1];
                }
                NOCONTA(&el,&nd);if(enExcepcion==1)return;
                
                //* Si pertenece
            }
            else if(codA[el-1][1-1] == 9)                  
            {
                //* Deslizamiento
                for( i=1; i<=3; i++)                        
                {
                    a[3*(nd-1)+i-1][3*(nelA+el)-2-1]=-BE[i-1][1-1]*Emedia;
                    for( j=1; j<=3; j++)                              
                    {
                        a[3*(nd-1)+i-1][3*(el-1)+j-1]=AE[i-1][j-1];
                    }
                }
            }
        }
    }
    
    //* Suma al termino independiente los coeficientes del problema termoelastico
    if(tpproTE == 1)      
    {
        for( nd=1; nd<=nelA; nd++)            
        {
            for( el=1; el<=nelA; el++)                  
            {
                int arTemp_8;int arTemp_9;int arTemp_10;arTemp_8 = 1-1; for(arTemp_9=el*3 -2;arTemp_9<=el*3;arTemp_9+=1)
                {
                    arTemp_8 += 1;for(arTemp_10=nd;arTemp_10<=nd;arTemp_10+=1)
                    {
                        CTE[arTemp_8-1] =  CTE_A[arTemp_9-1][arTemp_10-1];
                    }
                }
                ;
                int arTemp_11;int arTemp_12;int arTemp_13;arTemp_11 = 1-1; for(arTemp_12=el*3 -2;arTemp_12<=el*3;arTemp_12+=1)
                {
                    arTemp_11 += 1;for(arTemp_13=nd;arTemp_13<=nd;arTemp_13+=1)
                    {
                        DTTE[arTemp_11-1] =  DTE_A[arTemp_12-1][arTemp_13-1];
                    }
                }
                ;
                
                for( i=1; i<=3; i++)                        
                {
                    b[3*(nd-1)+i-1]=b[3*(nd-1)+i-1]+DTTE[i-1]*(tempA[el-1]-tref)-
                    CTE[i-1]*flujA[el-1];;
                }
            }
        }
    }
    //* Abre ficheros de coeficientes elasticos del cuerpo B
    //* Monta la parte correspondiente al cuerpo B del problema elastico
    printf("  Monta la parte correspondiente al cuerpo B del problema elastico\n");
    for( nd=1; nd<=nelB; nd++)      
    {
        for( el=1; el<=nelB; el++)            
        {
            //* Lee los coeficientes
            int arTemp_14;int arTemp_15;int arTemp_16;int arTemp_17;arTemp_16 = el*3 -2-1; for(arTemp_14=1;arTemp_14<=3;arTemp_14+=1)
            {
                arTemp_16 += 1;arTemp_17 = nd*3 -2-1; for(arTemp_15=1;arTemp_15<=3;arTemp_15+=1)
                {
                    arTemp_17 += 1;AE[arTemp_14-1][arTemp_15-1] =  AE_B[arTemp_16-1][arTemp_17-1];
                }
            }
            ;
            int arTemp_18;int arTemp_19;int arTemp_20;int arTemp_21;arTemp_20 = el*3 -2-1; for(arTemp_18=1;arTemp_18<=3;arTemp_18+=1)
            {
                arTemp_20 += 1;arTemp_21 = nd*3 -2-1; for(arTemp_19=1;arTemp_19<=3;arTemp_19+=1)
                {
                    arTemp_21 += 1;BE[arTemp_18-1][arTemp_19-1] =  BE_B[arTemp_20-1][arTemp_21-1];
                }
            }
            ;
            
            //* Mira si el nodo el pertenace a la zona actual de contacto
            if(codB[el-1][1-1] < 9)                  
            {
                //* No pertenece
                nel=nelA;
                cod=codB[el-1][1-1];
                for( m=1; m<=6; m++)                        
                {
                    cc[m-1]=ccB[el-1][m-1];
                }
                NOCONTA(&el,&nd);if(enExcepcion==1)return;
                //* Si pertenece
            }
            else if(codB[el-1][1-1] == 9)                  
            {
                //* Deslizamiento
                for( i=1; i<=3; i++)                        
                {
                    for( j=2; j<=3; j++)                              
                    {
                        a[3*(nelA+nd-1)+i-1][3*(nelA+el-1)+j-1]=AE[i-1][j-1];
                    }
                    a[3*(nelA+nd-1)+i-1][3*(nelA+el)-2-1]=-BE[i-1][1-1]*Emedia;
                    a[3*(nelA+nd-1)+i-1][3*el-2-1]=-AE[i-1][1-1];
                    b[3*(nelA+nd-1)+i-1]=b[3*(nelA+nd-1)+i-1]-gap[el-1]*AE[i-1][1-1];
                }
            }
        }
    }
    
    //* Suma al termino independiente los coeficientes del problema termoelastico
    if(tpproTE == 1)      
    {
        for( nd=1; nd<=nelB; nd++)            
        {
            for( el=1; el<=nelB; el++)                  
            {
                int arTemp_22;int arTemp_23;int arTemp_24;arTemp_22 = 1-1; for(arTemp_23=el*3 -2;arTemp_23<=el*3;arTemp_23+=1)
                {
                    arTemp_22 += 1;for(arTemp_24=nd;arTemp_24<=nd;arTemp_24+=1)
                    {
                        CTE[arTemp_22-1] =  CTE_B[arTemp_23-1][arTemp_24-1];
                    }
                }
                ;
                int arTemp_25;int arTemp_26;int arTemp_27;arTemp_25 = 1-1; for(arTemp_26=el*3 -2;arTemp_26<=el*3;arTemp_26+=1)
                {
                    arTemp_25 += 1;for(arTemp_27=nd;arTemp_27<=nd;arTemp_27+=1)
                    {
                        DTTE[arTemp_25-1] =  DTE_B[arTemp_26-1][arTemp_27-1];
                    }
                }
                ;
                
                for( i=1; i<=3; i++)                        
                {
                    b[3*(nelA+nd-1)+i-1]=b[3*(nelA+nd-1)+i-1]+DTTE[i-1]*(tempB[el-1]-tref)-
                    CTE[i-1]*flujB[el-1];;
                }
            }
        }
    }
    printf("fin montaje elas\n");
    
    return;
}


void INTERPR()
{
    //* Declaracion de variables
    
    int j,k;    //auxiliar
    int ix; //parametro para identificar la numeración de las variables
    //* Interpreta resultados del cuerpo A
    for( j=1; j<=nelA; j++)      
    {
        //* Mira si el nodo j perteneca a la zona de contacto actual
        if(codA[j-1][1-1] < 9)            
        {
            //* No pertenece
            ix=0.0;
            INTLIBR(&j,&ix);if(enExcepcion==1)return;
            //* Si pertenece
        }
        else if(codA[j-1][1-1] == 9)            
        {
            //* Deslizamiento
            for( k=1; k<=3; k++)                  
            {
                ccA[j-1][k-1]=x[3*(j-1)+k-1];
            }
            ccA[j-1][4-1]=x[3*(j+nelA)-2-1]*Emedia;
            ccA[j-1][5-1]=0.;
            ccA[j-1][6-1]=0.;
        }
        else            
        {
            //* Adhesion
            for( k=1; k<=3; k++)                  
            {
                ccA[j-1][k-1]=x[3*(j-1)+k-1];
                ccA[j-1][k+3-1]=x[3*(j+nelA-1)+k-1]*Emedia;
            }
        }
    }
    //* Interpreta resultados del cuerpo B
    for( j=1; j<=nelB; j++)      
    {
        //* Mira si el nodo j perteneca a la zona de contacto actual
        if(codB[j-1][1-1] < 9)            
        {
            //* No pertenece
            ix=3*nelA;
            INTLIBR(&j,&ix);if(enExcepcion==1)return;
            //* Si pertenece
        }
        else if(codB[j-1][1-1] == 9)            
        {
            //* Deslizamiento
            ccB[j-1][1-1]=gap[j-1]-ccA[j-1][1-1];
            ccB[j-1][4-1]=ccA[j-1][4-1];
            ccB[j-1][5-1]=-ccA[j-1][5-1];
            ccB[j-1][6-1]=ccA[j-1][6-1];
            for( k=2; k<=3; k++)                  
            {
                ccB[j-1][k-1]=x[3*(nelA+j-1)+k-1];
            }
        }
        else            
        {
            //* Adhesion
            ccB[j-1][1-1]=gap[j-1]-ccA[j-1][1-1];
            ccB[j-1][2-1]=ccA[j-1][2-1];
            ccB[j-1][3-1]=-ccA[j-1][3-1];
            ccB[j-1][4-1]=ccA[j-1][4-1];
            ccB[j-1][5-1]=-ccA[j-1][5-1];
            ccB[j-1][6-1]=ccA[j-1][6-1];
        }
    }
    return;
}


void DESLIZA()
{
    //* Declaracion de variables
    double tang;
    int i;  //auxiliar
    for( i=1; i<=nelpc; i++)      
    {
        if(codA[i-1][1-1] > 8)            
        {
            ve2[i-1]=ccA[i-1][2-1]-ccB[i-1][2-1];
            ve3[i-1]=ccA[i-1][3-1]+ccB[i-1][3-1];
            ve[i-1]=sqrt(pow(ve2[i-1],2)+pow(ve3[i-1],2));
            if(ve2[i-1] != 0. )                  
            {
                tang=ve3[i-1]/ve2[i-1];
                anguv[i-1]=atan(tang);
                if(ve2[i-1] < 0. )anguv[i-1]=anguv[i-1]+4.0*atan(1.0);              //3.141592654
            }
            else                  
            {
                anguv[i-1]=4.0*atan(1.0)/2.;
                if(ve3[i-1] < 0. )anguv[i-1]=anguv[i-1]+4.0*atan(1.0);              //3.141592654
            }
        }
        else            
        {
            ve[i-1]=0.;
            ve2[i-1]=0.;
            ve3[i-1]=0.;
            anguv[i-1]=0.;
        }
        anguv[i-1]=anguv[i-1]*180./4.0*atan(1.0);       //3.141592654
    }
    return;
}



void CHEQUEO()
{
    //* Declaracion de variables
    //* Inicializa los chivatos
    ifla=0;
    //* Chequeo de tracciones
    CHETRAC();if(enExcepcion==1)return;
    printf("numero de elementos que salen de la zona de contacto:%d\n",ifla);
    //* Si es negativo, retorna
    if(ifla != 0)return;
    //* Chequeo de interpenetraciones
    CHEINTE();if(enExcepcion==1)return;
    printf("numero de elementos que entran en la zona de contacto:%d\n",ifla);
    return;
}



void COMPRESIS()
{
    //* Declaracion de variables
    
    
    int nd; //auxiliar
    double mc,sigc; //Parametros de rugosidades
    double Hmicro;  //Parametros de microdurezas
    double kc,hc,hj,hb,hg;  //conductividades y conductancias contacto
    double Y,Mgas;  //interfase
    double tempg;   //Temperatura media del gas
    //* Ponemos a cero el chivato chires
    chires=0;
    //* Calculo de la resistencia térmica
    
    for( nd=1; nd<= nelpc     ; nd++)        
    {
        if(codA[nd-1][2-1] == 3)            
        {
            //*calcula parámetros de contacto DATOS: k en W/mmºC, sig en mm, Hmicro en MPa
            kc=2*lamA*lamB/(lamA+lamB);
            Hmicro=2*HAmicro*HBmicro/(HAmicro+HBmicro);
            mc=pow((pow(mA,2)+pow(mB,2)),0.50);
            sigc=pow((pow(sigA,2)+pow(sigB,2)),0.50);
            //* Calcula resistencia térmica de contacto
            
            //Ley de Yovanovich
            hc=1.25*kc*mc/sigc*pow((fabs(ccA[nd-1][4-1])/Hmicro),0.950);
            //Ley de Pilar
            //_____________________________________________________
            //hc=1.45*kc*mc/sigc*(abs(ccA(nd,4))/Hmicro)**0.985D0
            //_____________________________________________________
            
            if(TIM == 1)                
            {
                hg=0.00;
            }
            else                
            {
                Y=1.53*sigc*pow((fabs(ccA[nd-1][4-1])/Hmicro),(-0.0970));
                if(TIM == 2)                    
                {
                    tempg=(tempA[nd-1]+tempB[nd-1])/2;
                    Mgas=M0*(tempg/50)*(0.1013000/(fabs(ccA[nd-1][4-1])));
                }
                else                    
                {
                    Mgas=0.0;
                }
                hg=kg/(Y+Mgas);
            }
            hj=hc+hg;
            if(TIM == 4)                
            {
                hb=kg/(tTIM*(1-fabs(ccA[nd-1][4-1])/ETIM));
                //hb=kg/tTIM
                RTCC[nd-1]=1/hj+1/hb;
            }
            else                
            {
                RTCC[nd-1]=1/hj;
            }
            //* Resistencia ejemplo Kishimoto
            //RTCC(nd)=0.002*exp(-0.01*(abs(ccA(nd,4))))*1e6
        }
    }
    //* Compara resistencias termicas nodos en contacto
    
    for( nd=1; nd<= nelpc ; nd++)            
    {
        if(codA[nd-1][2-1] == 3)                
        {
            if(fabs((RTC[nd-1]-RTCC[nd-1])*100/RTCC[nd-1]) > (10.0) )                  
            {
                chires=1;
                goto l3434;
            }
        }
    }
    //* Asigna valores de resistencia termica
    l3434:      for( nd=1; nd<= nelpc; nd++)    
    {
        if(codA[nd-1][2-1] == 3)            
        {
            RTC[nd-1]=RTCC[nd-1];
        }
    }
    return;
}



void SALIDA_SOLU()
{
    //* Declaracion de variables
    int nd; //auxiliar
    
    //* abre ficheros de salida
    out3s = fopen(nomArchSalElas, "w");
    
    fprintf(out3s," %s",titulo);    // Formato: 5
    fprintf(out3s," Resultados problema ELÁSTICO de contacto\n");   // Formato: 10
    fprintf(out3s," \n");   // Formato: 20
    fprintf(out3s," DESPLAZAMIENTOS Y TENSIONES\n");    // Formato: 30
    fprintf(out3s," \n");   // Formato: 20
    
    //* CUERPO A
    
    fprintf(out3s," ==================== CUERPO A ====================\n"); // Formato: 35
    fprintf(out3s,"  - Zona de contacto -\n");  // Formato: 40
    
    //*  RESULTADOS EN: Movimientos [µm], Presiones [MPa], Resistencias en [mm2 ºC/W]
    //*  Zona de contacto de A
    
    for( nd=1; nd<=nelpc; nd++)    
    {
        fprintf(out3s," %4d %12.5f %12.4f %12.4f %12.4f %12.4f %12.4f %13.6f\n",nd,ccA[nd-1][1-1]*1e3,ccA[nd-1][2-1],ccA[nd-1][3-1],-ccA[nd-1][4-1],ccA[nd-1][5-1],ccA[nd-1][6-1],RTC[nd-1]);       // Formato: 100
    }
    //*  Zona libre de A
    fprintf(out3s,"  - Zona libre -\n");    // Formato: 60
    
    for( nd=nelpc+1; nd<=nelA; nd++)    
    {
        fprintf(out3s," %4d %12.5f %12.4f %12.4f %12.4f %12.4f %12.4f \n",nd,ccA[nd-1][1-1]*1e3,ccA[nd-1][2-1],ccA[nd-1][3-1],-ccA[nd-1][4-1],ccA[nd-1][5-1],ccA[nd-1][6-1]);       // Formato: 110
    }
    fprintf(out3s," \n");   // Formato: 20
    
    //* CUERPO B
    fprintf(out3s," ==================== CUERPO B ====================\n"); // Formato: 55
    fprintf(out3s,"  - Zona de contacto -\n");  // Formato: 40
    
    
    //*  Zona de contacto de B
    
    for( nd=1; nd<=nelpc; nd++)        
    {
        fprintf(out3s," %4d %12.5f %12.4f %12.4f %12.4f %12.4f %12.4f %13.6f\n",nd,ccB[nd-1][1-1]*1e3,ccB[nd-1][2-1],ccB[nd-1][3-1],-ccB[nd-1][4-1],ccB[nd-1][5-1],ccB[nd-1][6-1],RTC[nd-1]);       // Formato: 100
    }
    //*  Zona libre de B
    fprintf(out3s,"  - Zona libre -\n");    // Formato: 60
    //write(out3s,56)
    for( nd=nelpc+1; nd<=nelB; nd++)        
    {
        fprintf(out3s," %4d %12.5f %12.4f %12.4f %12.4f %12.4f %12.4f \n",nd,ccB[nd-1][1-1]*1e3,ccB[nd-1][2-1],ccB[nd-1][3-1],-ccB[nd-1][4-1],ccB[nd-1][5-1],ccB[nd-1][6-1]);       // Formato: 110
    }
    //*  Cierra fichero
    
    fclose(out3s);
    
    //* Formatos
    return;
}


void inicializar(EntradaCuerpo* entradaCuerpo, TIPO_ENTRADA tipoEntrada)
{
    entradaCuerpo->tipoEntrada = tipoEntrada;

    entradaCuerpo->exT  = (double **)generarMatriz(sizeof(double), 5000, 3, tipoEntrada);
    entradaCuerpo->conT = (int **)generarMatriz(sizeof(int), 5000, 3, tipoEntrada);
    entradaCuerpo->ndT  = (double **)generarMatriz(sizeof(double), 5000, 3, tipoEntrada);
    entradaCuerpo->locT = (double **)generarMatriz(sizeof(double), 5000, 9, tipoEntrada);

    entradaCuerpo->AE   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT * 3, tipoEntrada);
    entradaCuerpo->BE   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT * 3, tipoEntrada);
    entradaCuerpo->AT   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT, entradaCuerpo->nelT, tipoEntrada);
    entradaCuerpo->BT   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT, entradaCuerpo->nelT, tipoEntrada);
    entradaCuerpo->CTE  = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT, tipoEntrada);
    entradaCuerpo->DTE  = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT, tipoEntrada);
}

void** generarMatriz(int tamanioElemento, int filas, int columnas, TIPO_ENTRADA tipoEntrada)
{
    void** variable;

    switch (tipoEntrada)
    {
        case HOST:
            variable = (void **)alloc2DOnHost(filas, columnas, tamanioElemento);
            break;

        case CUDA:
            variable = (void **)alloc2DOnDevice(filas, columnas, tamanioElemento);
            break;
    }

    return variable;
}

/**
 * Allocates a 2 dimensional array.
 *
 * Its elements are initialized to 0 by default.
 * It can be freed with a single free() call.
 * It is possible to access this array as array[n][m] in the code.
 * If you want a deep explanation, please, email me at
 * ipazce00@estudiantes.unileon.es
 *
 * @param rows  number of rows.
 * @param columns number of columns.
 * @param sizeOfElement size in bytes of each element.
 *
 * @return a 2 dimensional array.
 */
void** alloc2DOnHost(int rows, int columns, int sizeOfElement)
{
    int header = rows * sizeof(void*);
    int body = rows * columns * sizeOfElement;

    void** rowptr = (void**)malloc(header + body);
    char* values = (char*)(rowptr + rows);

    int index=0;
    for (int i=0; i< rows*columns * sizeOfElement; i++)
    {
        values[i] = 0;
        if (i % (sizeof(int)*columns) == 0)
            rowptr[index++] = (void*)&values[i];
    }


    return rowptr;
}

__global__ void __organize2DMatrix(void** matrix, int rows, int cols, int sizeOfElement)
{
    char* values = (char*)(matrix + rows);

    int index=0;
    for (int i=0; i< rows*cols * sizeOfElement; i++)
    {
        values[i] = 0;
        if (i % (sizeof(int)*cols) == 0)
            matrix[index++] = (void*)&values[i];
    }
}

void ** alloc2DOnDevice(int rows, int cols, int sizeOfElement)
{
    int header = rows * sizeof(void*);
    int body = rows * cols * sizeOfElement;

    void** rowptr;
    hipMalloc(&rowptr, header+body);

    __organize2DMatrix<<<1,1>>>(rowptr, rows, cols, sizeOfElement);

    hipDeviceSynchronize();

    return rowptr;
}



void liberarMatriz(void** matriz, TIPO_ENTRADA tipoEntrada)
{
    switch (tipoEntrada)
    {
        case HOST:
            free(matriz);
            break;

        case CUDA:
            hipFree(matriz);
            break;
    }
}

void finalizar(EntradaCuerpo* entradaCuerpo)
{
    liberarMatriz((void**)entradaCuerpo->exT, entradaCuerpo->tipoEntrada);
    liberarMatriz((void**)entradaCuerpo->conT, entradaCuerpo->tipoEntrada);
    liberarMatriz((void**)entradaCuerpo->ndT, entradaCuerpo->tipoEntrada);
    liberarMatriz((void**)entradaCuerpo->locT, entradaCuerpo->tipoEntrada);

    liberarMatriz((void**)entradaCuerpo->AE, entradaCuerpo->tipoEntrada);
    liberarMatriz((void**)entradaCuerpo->BE, entradaCuerpo->tipoEntrada);
    liberarMatriz((void**)entradaCuerpo->AT, entradaCuerpo->tipoEntrada);
    liberarMatriz((void**)entradaCuerpo->BT, entradaCuerpo->tipoEntrada);
    liberarMatriz((void**)entradaCuerpo->CTE, entradaCuerpo->tipoEntrada);
    liberarMatriz((void**)entradaCuerpo->DTE, entradaCuerpo->tipoEntrada);
}

void copiarMatriz(void** matrizDestino, void** matrizOrigen, int tamanioElemento, int filas, int columnas, int direccion)
{

    switch (direccion)
    {
        case MEMCPY_HOST_TO_HOST:
            memcpy(matrizDestino+filas, matrizOrigen+filas,
                   filas*columnas*sizeof(tamanioElemento) - sizeof(void*)*filas);
            break;
        case hipMemcpyDeviceToHost:
            hipMemcpy(matrizDestino+filas, matrizOrigen+filas,
                       filas*columnas*sizeof(tamanioElemento) - sizeof(void*)*filas, hipMemcpyDeviceToHost);
            break;
        case hipMemcpyHostToDevice:
            hipMemcpy(matrizDestino+filas, matrizOrigen+filas,
                       filas*columnas*sizeof(tamanioElemento) - sizeof(void*)*filas, hipMemcpyHostToDevice);
            break;
    }
}
