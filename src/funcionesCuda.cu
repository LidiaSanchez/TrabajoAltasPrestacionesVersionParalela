#include "hip/hip_runtime.h"
#include "funcionesCuda.h"

void inicializar(EntradaCuerpo* entradaCuerpo, CONTEXTO contexto)
{
    entradaCuerpo->contexto = contexto;

    entradaCuerpo->exT  = (double **)generarMatriz(sizeof(double), 5000, 3, contexto);
    entradaCuerpo->conT = (int **)generarMatriz(sizeof(int), 5000, 3, contexto);
    entradaCuerpo->ndT  = (double **)generarMatriz(sizeof(double), 5000, 3, contexto);
    entradaCuerpo->locT = (double **)generarMatriz(sizeof(double), 5000, 9, contexto);

    entradaCuerpo->AE_T   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT * 3, contexto);
    entradaCuerpo->BE_T   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT * 3, contexto);
    entradaCuerpo->AT_T   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT, entradaCuerpo->nelT, contexto);
    entradaCuerpo->BT_T   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT, entradaCuerpo->nelT, contexto);
    entradaCuerpo->CTE_T  = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT, contexto);
    entradaCuerpo->DTE_T  = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT, contexto);

    entradaCuerpo->AE     = (double **)generarMatriz(sizeof(double), 3, 3, contexto);
    entradaCuerpo->BE     = (double **)generarMatriz(sizeof(double), 3, 3, contexto);
    entradaCuerpo->CTE    = (double *)generarVector(sizeof(double), 3, contexto);
    entradaCuerpo->DTTE   = (double *)generarVector(sizeof(double), 3, contexto);
    entradaCuerpo->DTE    = (double **)generarMatriz(sizeof(double), 3, 3, contexto);

    entradaCuerpo->ndCol  = (double *)generarVector(sizeof(double), 3, contexto);
    entradaCuerpo->extr   = (double **)generarMatriz(sizeof(double), 4, 3, contexto);

}

void** generarMatriz(int tamanioElemento, int filas, int columnas, CONTEXTO contexto)
{
    void** variable;

    switch (contexto)
    {
        case HOST:
            variable = (void **)alloc2DOnHost(filas, columnas, tamanioElemento);
            break;

        case CUDA:
            variable = (void **)alloc2DOnDevice(filas, columnas, tamanioElemento);
            break;
    }

    return variable;
}

void* generarVector(int tamanioElemento, int longitud, CONTEXTO contexto)
{
    void* variable;

    switch (contexto)
    {
        case HOST:
            variable = (void*)malloc(tamanioElemento * longitud);
            break;

        case CUDA:
            hipMalloc(&variable, tamanioElemento * longitud);
            break;
    }

    return variable;
}

void** alloc2DOnHost(int rows, int columns, int sizeOfElement)
{
    int header = rows * sizeof(void*);
    int body = rows * columns * sizeOfElement;

    void** rowptr = (void**)malloc(header + body);
    char* values = (char*)(rowptr + rows);

    for (int i=0; i< rows*columns * sizeOfElement; i++)
        values[i] = 0;

    for (int i=0; i<rows; i++)
        rowptr[i] = (void*) &values[i*columns*sizeOfElement];

    return rowptr;
}

void ** alloc2DOnDevice(int rows, int cols, int sizeOfElement)
{
    int header = rows * sizeof(void*);
    int body = rows * cols * sizeOfElement;

    void** rowptr;

    hipMalloc(&rowptr, header+body);

    __organize2DMatrix<<<1,1>>>(rowptr, rows, cols, sizeOfElement);

    hipDeviceSynchronize();

    return rowptr;
}

__global__ void __organize2DMatrix(void** matrix, int rows, int cols, int sizeOfElement)
{
    char* values = (char*)(matrix + rows);

    for (int i=0; i<rows; i++)
        matrix[i] = (void*) &values[i*cols*sizeOfElement];
}

void copiarMatriz(void** matrizDestino, void** matrizOrigen, int tamanioElemento, int filas, int columnas,
                  int direccion)
{
    switch (direccion)
    {
        case MEMCPY_HOST_TO_HOST:
            memcpy(matrizDestino+filas, matrizOrigen+filas,
                   filas*columnas*tamanioElemento);
            break;
        case hipMemcpyDeviceToHost:
            hipMemcpy(matrizDestino+filas, matrizOrigen+filas,
                       filas*columnas*tamanioElemento, hipMemcpyDeviceToHost);
            break;
        case hipMemcpyHostToDevice:
            hipMemcpy(matrizDestino+filas, matrizOrigen+filas,
                       filas*columnas*tamanioElemento, hipMemcpyHostToDevice);
            break;
    }
}

void copiarVector(void* vectorDestino, void* vectorOrigen, int tamanioElemento, int longitud, int direccion)
{
    switch (direccion)
    {
        case MEMCPY_HOST_TO_HOST:
            memcpy(vectorDestino, vectorOrigen,
                   longitud*tamanioElemento);
            break;
        case hipMemcpyDeviceToHost:
            hipMemcpy(vectorDestino, vectorOrigen,
                       longitud*tamanioElemento, hipMemcpyDeviceToHost);
            break;
        case hipMemcpyHostToDevice:
            hipMemcpy(vectorDestino, vectorOrigen,
                       longitud*tamanioElemento, hipMemcpyHostToDevice);
            break;
    }
}

void liberarVector(void* vector, CONTEXTO contexto)
{
    switch (contexto)
    {
        case HOST:
            free(vector);
            break;

        case CUDA:
            hipFree(vector);
            break;
    }
}

void liberarMatriz(void** matriz, CONTEXTO contexto)
{
    switch (contexto)
    {
        case HOST:
            free(matriz);
            break;

        case CUDA:
            hipFree(matriz);
            break;
    }
}

void copiarEstructura(EntradaCuerpo* destino, EntradaCuerpo* origen, int direccion)
{
    destino->nexT = origen->nexT;
    destino->nelT = origen->nelT;
    destino->ET = origen->ET;
    destino->alT = origen->alT;
    destino->nuT = origen->nuT;
    destino->GT = origen->GT;

    destino->simXY = origen->simXY;// Flag de simetría respecto del plano xOy
    destino->simXZ = origen->simXZ;// Flag de simetría respecto del plano xOz
    destino->simYZ = origen->simYZ;// Flag de simetría respecto del plano yOz
    destino->tpproE = origen->tpproE;// Flag de tipo de problema elastico
    destino->tpproT = origen->tpproT;// Flag de tipo de problema termico
    destino->tpproTE = origen->tpproTE;// Flag de tipo de problema termoelastico
    destino->tpcarFP = origen->tpcarFP;// Flag de tipo de carga térmica. Fuentes puntuales
    destino->tpcarFL = origen->tpcarFL;// Flag de tipo de carga térmica. Fuentes lineales
    destino->tpcarFD = origen->tpcarFD;// Flag de tipo de carga térmica. Fuentes distribuidas
    destino->tpcarFC = origen->tpcarFC;// Flag de tipo de carga elástica. Fuerza centrífuga
    destino->tpcarPP = origen->tpcarPP;// Flag de tipo de carga elástica. Peso propio

    destino->cte1 = origen->cte1;// Constante en la integracion elastica
    destino->cte2 = origen->cte2;// Constante en la integracion elastica
    destino->cte3 = origen->cte3;// Constante en la integracion elastica
    destino->cte4 = origen->cte4;// Constante en la integracion termica
    destino->cte5 = origen->cte5;// Constante en la integracion termoelastica

    destino->reg = origen->reg;

    destino->enExcepcion = origen->enExcepcion;

    copiarMatriz((void**)destino->exT, (void**)origen->exT, sizeof(double), 5000, 3, direccion);
    copiarMatriz((void**)destino->conT, (void**)origen->conT, sizeof(int), 5000, 3, direccion);
    copiarMatriz((void**)destino->ndT, (void**)origen->ndT, sizeof(double), 5000, 3, direccion);
    copiarMatriz((void**)destino->locT, (void**)origen->locT, sizeof(double), 5000, 9, direccion);

    copiarMatriz((void**)destino->AE_T, (void**)origen->AE_T, sizeof(double), origen->nelT * 3, origen->nelT * 3, direccion);
    copiarMatriz((void**)destino->BE_T, (void**)origen->BE_T, sizeof(double), origen->nelT * 3, origen->nelT * 3, direccion);
    copiarMatriz((void**)destino->AT_T, (void**)origen->AT_T, sizeof(double), origen->nelT, origen->nelT, direccion);
    copiarMatriz((void**)destino->BE_T, (void**)origen->BE_T, sizeof(double), origen->nelT, origen->nelT, direccion);
    copiarMatriz((void**)destino->CTE_T, (void**)origen->CTE_T, sizeof(double), origen->nelT * 3, origen->nelT, direccion);
    copiarMatriz((void**)destino->DTE_T, (void**)origen->DTE_T, sizeof(double), origen->nelT * 3, origen->nelT, direccion);

    copiarMatriz((void**)destino->AE, (void**)origen->AE, sizeof(double), 3, 3, direccion);
    copiarMatriz((void**)destino->BE, (void**)origen->BE, sizeof(double), 3, 3, direccion);
    copiarVector((void*)destino->CTE, (void*)origen->CTE, sizeof(double), 3, direccion);
    copiarVector((void*)destino->DTTE, (void*)origen->DTTE, sizeof(double), 3, direccion);
    copiarMatriz((void**)destino->DTE, (void**)origen->DTE, sizeof(double), 3, 3, direccion);

    copiarVector((void*)destino->ndCol, (void*)origen->ndCol, sizeof(double), 3, direccion);
    destino->intenum = origen->intenum;
    destino->tipoSimetria = origen->tipoSimetria;
    copiarMatriz((void**)destino->extr, (void**)origen->extr, sizeof(double), 4, 3, direccion);

}


EntradaCuerpo* pasarADispositivo(EntradaCuerpo* entradaCuerpo)
{
    EntradaCuerpo* resultado;
    hipMalloc(&resultado, sizeof(EntradaCuerpo));
    hipMemcpy(resultado, entradaCuerpo,
               sizeof(EntradaCuerpo), hipMemcpyHostToDevice);
    return resultado;
}

EntradaCuerpo  obtenerDesdeDispositivo(EntradaCuerpo* entradaCuerpoDeDispositivo)
{
    EntradaCuerpo resultado;
    hipMemcpy(&resultado, entradaCuerpoDeDispositivo,
               sizeof(EntradaCuerpo), hipMemcpyDeviceToHost);
    hipFree(entradaCuerpoDeDispositivo);
    return resultado;
}

void finalizar(EntradaCuerpo* entradaCuerpo)
{
    liberarMatriz((void**)entradaCuerpo->exT, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->conT, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->ndT, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->locT, entradaCuerpo->contexto);

    liberarMatriz((void**)entradaCuerpo->AE_T, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->BE_T, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->AT_T, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->BT_T, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->CTE_T, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->DTE_T, entradaCuerpo->contexto);

    liberarMatriz((void**)entradaCuerpo->AE, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->BE, entradaCuerpo->contexto);
    liberarVector((void*)entradaCuerpo->CTE, entradaCuerpo->contexto);
    liberarVector((void*)entradaCuerpo->DTTE, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->DTE, entradaCuerpo->contexto);

    liberarVector((void*)entradaCuerpo->ndCol, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->extr, entradaCuerpo->contexto);
}
