#include "hip/hip_runtime.h"
#include "funcionesMain.h"

int ObtenerSimetria(EntradaCuerpo* entradaCuerpo)
{
    // Buscamos la simetria
    int simetria;

    int&  simXY = entradaCuerpo->simXY;// Flag de simetría respecto del plano xOy
    int&  simXZ = entradaCuerpo->simXZ;// Flag de simetría respecto del plano xOz
    int&  simYZ = entradaCuerpo->simYZ;// Flag de simetría respecto del plano yOz


    if ((simXY != 1) && (simXZ != 1) && (simYZ != 1))
        simetria = NO_SIMETRIA_IMPLICITA;
    else if ((simXY == 1) && (simXZ != 1) && (simYZ != 1))
        simetria = SIMETRIA_PLANO_0XY;
    else if ((simXY != 1) && (simXZ == 1) && (simYZ != 1))
        simetria = SIMETRIA_PLANO_0XZ;
    else if ((simXY != 1) && (simXZ != 1) && (simYZ == 1))
        simetria = SIMETRIA_PLANO_0YZ;
    else if ((simXY == 1) && (simXZ == 1) && (simYZ != 1))
        simetria = SIMETRIA_PLANOS_0XY_0XZ;
    else if ((simXY == 1) && (simXZ != 1) && (simYZ == 1))
        simetria = SIMETRIA_PLANOS_0XY_0YZ;
    else if ((simXY != 1) && (simXZ == 1) && (simYZ == 1))
        simetria = SIMETRIA_PLANOS_0XZ_0YZ;
    else if ((simXY == 1) && (simXZ == 1) && (simYZ == 1))
        simetria = SIMETRIA_TRES_PLANOS_COORDENADOS;
    else
        simetria = SIMETRIA_ERRONEA;

    return simetria;
}


void CODIFICADA()
{
    //* Declaracion de variables
    
    
    int  bContacto; //Indica si es superficie de contacto
    int  nele,nex,nexpc,nin;    //Número de elementos y extremos actuales. Número de extremos en la zona de contacto
    int  ifla1; //Flag para continuar o no la lectura
    int  tipoS; //Tipo de superficie
    int  i,j;   //Auxiliares
    //* Lee los datos relativos a la zona de contacto
    nele=1;
    nex=1;
    ifla1=0;
    bContacto=1;
    while(ifla1 == 0)
    {
        leeEntero(in1,&tipoS);leeEntero(in1,&ifla1);leeLinea(in1);
        //* Segun el tipo de contorno sigue un camino u otro
        if(tipoS == 0)        
        {
            //* Contacto S4-S4
            SUPERFICIE_CUATRO(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else if(tipoS == 1)        
        {
            //* Contacto S3-S3
            SUPERFICIE_TRES(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else        
        {
            //* Contacto entre otros tipos de superficies
            printf(" **ERROR**: SUPERFICIE DE CONTACTO NO PROGRAMADA\n");
            printf("REVISE LA ENTRADA DE DATOS\n"); enExcepcion=1;return;
        }
    }
    //* Se ha terminado la zona de contacto
    nelpc=nele-1;
    nexpc=nex-1;
    //* Lee codigos de la zona de contacto
    nin=1;
    LEE_CODIGOS(&nin,&nelpc);if(enExcepcion==1)return;
    //* Inicializa condiciones de contorno de la zona de contacto
    for( i=1; i<=nelpc; i++)      
    {
        codA[i-1][1-1]=codB[i-1][1-1];
        codA[i-1][2-1]=codB[i-1][2-1];
        for( j=1; j<=10; j++)        
        {
            ccA[i-1][j-1]=0.;
            ccB[i-1][j-1]=0.;
        }
    }
    //* Lee los datos relativos a la zona libre de "A"
    ifla1=0;
    bContacto=0;
    while(ifla1 == 0)
    {
        leeEntero(in1,&tipoS);leeEntero(in1,&ifla1);leeLinea(in1);
        //* Segun el tipo de contorno sigue un camino u otro
        if(tipoS == 0)        
        {
            //* Superficie S4
            SUPERFICIE_CUATRO(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else if(tipoS == 1)        
        {
            //* Superficie S3-S3
            SUPERFICIE_TRES(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else        
        {
            //* Otros tipos de superficies
            printf(" **ERROR**: SUPERFICIE NO PROGRAMADA EN SOLIDO A\n");
            printf("REVISE LA ENTRADA DE DATOS\n"); enExcepcion=1;return;
        }
    }
    //* Se ha terminado la zona libre de "A"
    nelA=nele-1;
    nexA=nex-1;
    //* Lee codigos de la zona libre de A
    nin=nelpc+1;
    LEE_CODIGOS(&nin,&nelA);if(enExcepcion==1)return;
    //* Lee condiciones de contorno de la zona libre de A
    LEE_CC(&nin,&nelA);if(enExcepcion==1)return;
    //* Asigna valores a variables relativas a "A"
    for( i=nexpc+1; i<=nexA; i++)      
    {
        for( j=1; j<=3; j++)        
        {
            exA[i-1][j-1]=exB[i-1][j-1];
        }
    }
    for( i=nelpc+1; i<=nelA; i++)      
    {
        codA[i-1][1-1]=codB[i-1][1-1];
        codA[i-1][2-1]=codB[i-1][2-1];
        for( j=1; j<=3; j++)        
        {
            conA[i-1][j-1]=conB[i-1][j-1];
        }
        for( j=1; j<=10; j++)        
        {
            ccA[i-1][j-1]=ccB[i-1][j-1];
        }

        for( j=1; j<=9; j++)        
        {
            locA[i-1][j-1]=locB[i-1][j-1];
        }
    }
    
    
#ifdef DEBUG
    printf("[funcionesMain.c] locA address: %p\n", locA);
    printf("[funcionesMain.c] locA[0][0] value: %f\n", locA[0][0]);
#endif
    
    //*    WRITE(*,*) 'SOLIDO B'
    //* Lee los datos relativos a la zona libre de "B"
    nele=nelpc+1;
    nex=nexpc+1;
    ifla1=0;
    while(ifla1 == 0)
    {
        leeEntero(in1,&tipoS);leeEntero(in1,&ifla1);leeLinea(in1);
        //* Segun el tipo de contorno sigue un camino u otro
        if(tipoS == 0)        
        {
            //* Superficie S4
            SUPERFICIE_CUATRO(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else if(tipoS == 1)        
        {
            //* Superficie S3-S3
            SUPERFICIE_TRES(&nele,&nex,&bContacto);if(enExcepcion==1)return;
        }
        else        
        {
            //* Otros tipos de superficies
            printf(" **ERROR**: SUPERFICIE NO PROGRAMADA EN SOLIDO A\n");
            printf("REVISE LA ENTRADA DE DATOS\n"); enExcepcion=1;return;
        }
    }
    //* Se ha terminado la zona libre de "B"
    nelB=nele-1;
    nexB=nex-1;
    //* Lee codigos de la zona libre de B
    nin=nelpc+1;
    LEE_CODIGOS(&nin,&nelB);if(enExcepcion==1)return;
    //* Lee condiciones de contorno de la zona libre de B
    LEE_CC(&nin,&nelB);if(enExcepcion==1)return;
    //* Calcula los nodos de ambos cuerpos
    CALCNODOS();if(enExcepcion==1)return;
    //* Calcula el sistema local de coordenadas de ambos cuerpos
    SISTLOCAL();if(enExcepcion==1)return;
    return;
}



void SEMICODIFICADA()
{
    //* Declaracion de variables
    
    
    
    
    //*       INCOMPLETA
    
    
    return;
}



void NODOS_CARGADOS()
{
    //* Declaracion de variables
    
    
    int  nd;    //Auxiliares
    int  i;
    //* Para el solido A
    nelwA=0;
    for( nd=nelpc+1; nd<=nelA; nd++)      
    {
        for( i=1; i<=10; i++)       
        {
            if(ccA[nd-1][i-1] != 0.0 )         
            {
                nelwA=nelwA+1;
                break;
            }
        }
    }
    //* Para el solido B
    nelwB=0;
    for( nd=nelpc+1; nd<=nelB; nd++)      
    {
        for( i=1; i<=10; i++)       
        {
            if(ccB[nd-1][i-1] != 0.0 )         
            {
                nelwB=nelwB+1;
                break;
            }
        }
    }
    return;
}



void DIR_CONTACTO()
{
    //* Declaracion de variables
    
    
    int  nd;    //Auxiliares
    int  i;
    double  modulo;
    //* Para todos los elementos de la zona potencial de contacto
    for( nd=1; nd<=nelpc; nd++)      
    {
        modulo=0.0;
        for( i=1; i<=3; i++)        
        {
            modulo=pow(modulo+(locA[nd-1][i-1]-locB[nd-1][i-1]),2);
        }
        modulo=sqrt(modulo);
        if(modulo != 0.0 )        
        {
            locA[nd-1][1-1]=(locA[nd-1][1-1]-locB[nd-1][1-1])/modulo;
            locB[nd-1][1-1]=-locA[nd-1][1-1];
            locA[nd-1][2-1]=(locA[nd-1][2-1]-locB[nd-1][2-1])/modulo;
            locB[nd-1][2-1]=-locA[nd-1][2-1];
            locA[nd-1][3-1]=(locA[nd-1][3-1]-locB[nd-1][3-1])/modulo;
            locB[nd-1][3-1]=-locA[nd-1][3-1];
        }
    }
    return;
}



void GAP_INICIAL()
{
    //* Declaracion de variables
    
    int  nd;    //Auxiliares
    double  modulo;
    //* Para todos los elementos de la zona potencial de contacto
    for( nd=1; nd<=nelpc; nd++)      
    {
        //* Calcula el modulo para ver si es nulo
        modulo=sqrt(pow((ndA[nd-1][1-1]-ndB[nd-1][1-1]),2)+pow((ndA[nd-1][2-1]-ndB[nd-1][2-1]),2)+pow((ndA[nd-1][3-1]-ndB[nd-1][3-1]),2));
        if(modulo == 0. )        
        {
            //* Separacion nula
            gap[nd-1]=0.;
        }
        else        
        {
            //* Calcula la separacion sobre la direccion de contacto
            gap[nd-1]=(ndA[nd-1][1-1]-ndB[nd-1][1-1])*locA[nd-1][1-1]+(ndA[nd-1][2-1]-ndB[nd-1][2-1])*locA[nd-1][2-1]+(ndA[nd-1][3-1]-ndB[nd-1][3-1])*locA[nd-1][3-1];
        }
    }
    return;
}




void INTEGRAL()
{
    //* Declaracion de variables
    
    
    int  el,ex; //Auxiliares
    int  j;
    float tiniA,tiniB,tfinA,tfinB;
    
    out11=81;
    out13=83;
    out14=84;

    // Asignamos valores mínimos del cuerpo A para poder inicializarlo
    cuerpoA[HOST].simXY = simXY;// Flag de simetría respecto del plano xOy
    cuerpoA[HOST].simXZ = simXZ;// Flag de simetría respecto del plano xOz
    cuerpoA[HOST].simYZ = simYZ;// Flag de simetría respecto del plano yOz
    cuerpoA[HOST].tpproE = tpproE;// Flag de tipo de problema elastico
    cuerpoA[HOST].tpproT = tpproT;// Flag de tipo de problema termico
    cuerpoA[HOST].tpproTE = tpproTE;// Flag de tipo de problema termoelastico
    cuerpoA[HOST].tpcarFP = tpcarFP;// Flag de tipo de carga térmica. Fuentes puntuales
    cuerpoA[HOST].tpcarFL = tpcarFL;// Flag de tipo de carga térmica. Fuentes lineales
    cuerpoA[HOST].tpcarFD = tpcarFD;// Flag de tipo de carga térmica. Fuentes distribuidas
    cuerpoA[HOST].tpcarFC = tpcarFC;// Flag de tipo de carga elástica. Fuerza centrífuga
    cuerpoA[HOST].tpcarPP = tpcarPP;// Flag de tipo de carga elástica. Peso propio
    cuerpoA[HOST].tipoSimetria = ObtenerSimetria(&cuerpoA[HOST]);


    // Calcula constantes
    cte1=16.0*4.0*atan(1.0)*GT*(1.0-nuT);
    cte2=1.0-2.0*nuT;
    cte3=8.0*(1.0-nuT)*4.0*atan(1.0);
    cte4=4.0*4.0*atan(1.0);
    cte5=alT*(1.0+nuT)/(8.0*4.0*atan(1.0)*(1.0-nuT));


    cuerpoA[HOST].cte1 = cte1;// Constante en la integracion elastica
    cuerpoA[HOST].cte2 = cte2;// Constante en la integracion elastica
    cuerpoA[HOST].cte3 = cte3;// Constante en la integracion elastica
    cuerpoA[HOST].cte4 = cte4;// Constante en la integracion termica
    cuerpoA[HOST].cte5 = cte5;

    cuerpoA[HOST].reg = reg;

    cuerpoA[HOST].enExcepcion = enExcepcion;

    cuerpoA[HOST].nexT = nexA;
    cuerpoA[HOST].nelT = nelA;
    cuerpoA[HOST].ET = EA;
    cuerpoA[HOST].alT = alA;
    cuerpoA[HOST].nuT = nuA;
    cuerpoA[HOST].GT = EA/(2.0*(1.0+nuA));

    cuerpoA[HOST].AT = AT;
    cuerpoA[HOST].BT = BT;

    cuerpoA[CUDA] = cuerpoA[HOST];


    // Asignamos valores mínimos del cuerpo B para poder inicializarlo
    cuerpoB[HOST].simXY = simXY;// Flag de simetría respecto del plano xOy
    cuerpoB[HOST].simXZ = simXZ;// Flag de simetría respecto del plano xOz
    cuerpoB[HOST].simYZ = simYZ;// Flag de simetría respecto del plano yOz
    cuerpoB[HOST].tpproE = tpproE;// Flag de tipo de problema elastico
    cuerpoB[HOST].tpproT = tpproT;// Flag de tipo de problema termico
    cuerpoB[HOST].tpproTE = tpproTE;// Flag de tipo de problema termoelastico
    cuerpoB[HOST].tpcarFP = tpcarFP;// Flag de tipo de carga térmica. Fuentes puntuales
    cuerpoB[HOST].tpcarFL = tpcarFL;// Flag de tipo de carga térmica. Fuentes lineales
    cuerpoB[HOST].tpcarFD = tpcarFD;// Flag de tipo de carga térmica. Fuentes distribuidas
    cuerpoB[HOST].tpcarFC = tpcarFC;// Flag de tipo de carga elástica. Fuerza centrífuga
    cuerpoB[HOST].tpcarPP = tpcarPP;// Flag de tipo de carga elástica. Peso propio
    cuerpoB[HOST].tipoSimetria = ObtenerSimetria(&cuerpoB[HOST]);

    cuerpoB[HOST].cte1 = cte1;// Constante en la integracion elastica
    cuerpoB[HOST].cte2 = cte2;// Constante en la integracion elastica
    cuerpoB[HOST].cte3 = cte3;// Constante en la integracion elastica
    cuerpoB[HOST].cte4 = cte4;// Constante en la integracion termica
    cuerpoB[HOST].cte5 = cte5;

    cuerpoB[HOST].reg = reg;

    cuerpoB[HOST].enExcepcion = enExcepcion;
    cuerpoB[HOST].nexT = nexB;
    cuerpoB[HOST].nelT = nelB;
    cuerpoB[HOST].ET = EB;
    cuerpoB[HOST].alT = alB;
    cuerpoB[HOST].nuT = nuB;
    cuerpoB[HOST].GT = EB/(2.0*(1.0+nuB));

    cuerpoB[HOST].AT = AT;
    cuerpoB[HOST].BT = BT;

    cuerpoB[CUDA] = cuerpoB[HOST];

    // Inicializamos las estructuras de entrada en cada contexto
    printf("Reservando memoria para estructuras de entrada de cuerpo en Host y Cuda... ");
    inicializar(&cuerpoA[HOST], HOST);
    inicializar(&cuerpoB[HOST], HOST);
    inicializar(&cuerpoA[CUDA], CUDA);
    inicializar(&cuerpoB[CUDA], CUDA);
    printf("Hecho.\n");

    for (int i=0; i<3; i++)
    {
        for(int j=0;j<3; j++)
        {
            cuerpoA[HOST].AE[i][j] = AE[i][j];
            cuerpoA[HOST].BE[i][j] = BE[i][j];
            cuerpoA[HOST].DTE[i][j] = DTE[i][j];
            cuerpoB[HOST].AE[i][j] = AE[i][j];
            cuerpoB[HOST].BE[i][j] = BE[i][j];
            cuerpoB[HOST].DTE[i][j] = DTE[i][j];
        }
        cuerpoA[HOST].CTE[i] = CTE[i];
        cuerpoA[HOST].DTTE[i] = DTTE[i];
        cuerpoB[HOST].CTE[i] = CTE[i];
        cuerpoB[HOST].DTTE[i] = DTTE[i];
    }
    for( ex=1; ex<=nexA; ex++)
        for( j=1; j<=3; j++)
            cuerpoA[HOST].exT[ex-1][j-1]=exA[ex-1][j-1];
    for( ex=1; ex<=nexB; ex++)
        for( j=1; j<=3; j++)
            cuerpoB[HOST].exT[ex-1][j-1]=exB[ex-1][j-1];
    for( el=1; el<=nelA; el++)
    {
        for( j=1; j<=3; j++)
        {
            cuerpoA[HOST].conT[el-1][j-1]=conA[el-1][j-1];
            cuerpoA[HOST].ndT[el-1][j-1]=ndA[el-1][j-1];
        }
        for( j=1; j<=9; j++)
            cuerpoA[HOST].locT[el-1][j-1]=locA[el-1][j-1];
    }
    for( el=1; el<=nelB; el++)
    {
        for( j=1; j<=3; j++)
        {
            cuerpoB[HOST].conT[el-1][j-1]=conB[el-1][j-1];
            cuerpoB[HOST].ndT[el-1][j-1]=ndB[el-1][j-1];
        }
        for( j=1; j<=9; j++)
            cuerpoB[HOST].locT[el-1][j-1]=locB[el-1][j-1];
    }

    //Vinculamos variables globales con los del cuerpo del host. Por si acaso.
    AE_A = cuerpoA[HOST].AE_T;
    BE_A = cuerpoA[HOST].BE_T;
    AT_A = cuerpoA[HOST].AT_T;
    BT_A = cuerpoA[HOST].BT_T;
    CTE_A = cuerpoA[HOST].CTE_T;
    DTE_A = cuerpoA[HOST].DTE_T;

    AE_B = cuerpoB[HOST].AE_T;
    BE_B = cuerpoB[HOST].BE_T;
    AT_B = cuerpoB[HOST].AT_T;
    BT_B = cuerpoB[HOST].BT_T;
    CTE_B = cuerpoB[HOST].CTE_T;
    DTE_B = cuerpoB[HOST].DTE_T;

    printf("==================== Entrando en INTEGRAL ====================\n");
    //* Lee los datos del problema
    

    //* Abre ficheros
    
    //* Calcula coeficientes del cuerpo "A"
    printf("++++++++ Calcula coeficientes del cuerpo \"A\" ++++++++\n");
    tiniA = clock()/CLOCKS_PER_SEC;
    printf("[%.3f] Inicio calculo coeficientes.\n",tiniA);


    copiarEstructura(&cuerpoA[CUDA], &cuerpoA[HOST], hipMemcpyHostToDevice);

    EntradaCuerpo* entradaCuerpoEnDispositivo = pasarADispositivo(&cuerpoA[CUDA]);

    host_printCheck_conT(cuerpoA[HOST].conT);
    host_printCheck_exT(cuerpoA[HOST].exT);
    //chivato<<<1,1>>>(entradaCuerpoEnDispositivo);
    COEFICIENTES<<<1,1>>>(entradaCuerpoEnDispositivo);
    hipDeviceSynchronize();
    printf("Error: %s\n",hipGetErrorString(hipGetLastError()));
    //COEFICIENTES<<<1, 1>>>(entradaCuerpoEnDispositivo);
    //hipDeviceSynchronize();
    cuerpoA[CUDA] = obtenerDesdeDispositivo(entradaCuerpoEnDispositivo);

    copiarEstructura(&cuerpoA[HOST], &cuerpoA[CUDA], hipMemcpyDeviceToHost);

    printf("Valor de cte5: %.2f", cuerpoA[HOST].cte5);
    if(cuerpoA[HOST].enExcepcion==1)return;
    tfinA = clock()/CLOCKS_PER_SEC;
    printf("[%.3f] Final calculo coeficientes.\n",tfinA);
    printf("\tTiempo Total= %f segundos\n",tfinA-tiniA);
    
    //* Cierra ficheros
    
    //* Asigna datos de entrada del cuerpo "B"

    //* Abre ficheros
    
    //* Calcula coeficientes del cuerpo "B"
    printf("+++++++ Calcula coeficientes del cuerpo \"B\" +++++++\n");
    tiniB = clock()/CLOCKS_PER_SEC;
    printf("[%.3f] Inicio calculo coeficientes.\n",tiniB);


    copiarEstructura(&cuerpoB[CUDA], &cuerpoB[HOST], hipMemcpyHostToDevice);
    entradaCuerpoEnDispositivo = pasarADispositivo(&cuerpoB[CUDA]);
    COEFICIENTES<<<1, 1>>>(entradaCuerpoEnDispositivo);if(enExcepcion==1)return;
    cuerpoB[CUDA] = obtenerDesdeDispositivo(entradaCuerpoEnDispositivo);

    copiarEstructura(&cuerpoB[HOST], &cuerpoB[CUDA], hipMemcpyDeviceToHost);

    if(cuerpoB[HOST].enExcepcion==1)return;
    //* Cierra ficheros
    
    //* Final del programa
    
    tfinB = clock()/CLOCKS_PER_SEC;
    printf("[%.3f] Final calculo coeficientes.\n",tfinB);
    printf("\tTiempo Total= %f segundos\n",tfinB-tiniB);
    
    printf("==================== Saliendo de INTEGRAL ===================\n");
    printf("TIEMPOOOOOOOOO: %f\n",(tfinA-tiniA)+(tfinB-tiniB));

    // Volcamos estructuras en variables globales para que pueda continuar el proceso.
    for (int i=0; i<3; i++)
    {
        for(int j=0;j<3; j++)
        {
            AE[i][j] = cuerpoA[HOST].AE[i][j];
            BE[i][j] = cuerpoA[HOST].BE[i][j];
            DTE[i][j] = cuerpoA[HOST].DTE[i][j];
            AE[i][j] = cuerpoB[HOST].AE[i][j];
            BE[i][j] = cuerpoB[HOST].BE[i][j];
            DTE[i][j] = cuerpoB[HOST].DTE[i][j];
        }
        CTE[i] = cuerpoA[HOST].CTE[i];
        DTTE[i] = cuerpoA[HOST].DTTE[i];
        CTE[i] = cuerpoB[HOST].CTE[i];
        DTTE[i] = cuerpoB[HOST].DTTE[i];
    }
    AT = cuerpoB->AT;
    BT = cuerpoB->BT;
    nexT = cuerpoB->nexT;
    nelT = cuerpoB->nelT;
    ET = cuerpoB->ET;
    alT = cuerpoB->alT;
    nuT = cuerpoB->nuT;
    GT = cuerpoB->GT;

    for( ex=1; ex<=nexB; ex++)
        for( j=1; j<=3; j++)
            exT[ex-1][j-1]=cuerpoB[HOST].exT[ex-1][j-1];

    for( el=1; el<=nelB; el++)
    {
        for( j=1; j<=3; j++)
        {
            conT[el-1][j-1]=cuerpoB[HOST].conT[el-1][j-1];
            ndT[el-1][j-1]=cuerpoB[HOST].ndT[el-1][j-1];
        }
        for( j=1; j<=9; j++)
            locT[el-1][j-1]=cuerpoB[HOST].locT[el-1][j-1];
    }
}


void TERMICO ()
{
    //* Declaracion de variables    
    //* Inicializa variables
    
    
    //* Comienzo del programa
    printf(" ==================== Entrando en TERMICO ====================\n");
    
    //* Se llama a la subrutina que trae los coeficientes   y monta el sistema
    MONTAJETER();if(enExcepcion==1)return;
    
    //* resuelve el sistema por el metodo de GAUSS
    GAUSS_SOLU();if(enExcepcion==1)return;
    //* Interpretacion de resultados (condiciones de contorno y contacto)
    CONTORNOTER();if(enExcepcion==1)return;
    //* Salida de resultados a ficheros
    SALIDATER();if(enExcepcion==1)return;
    //* Final del programa
    printf(" ==================== Saliendo de TERMICO ====================\n");
    return;
}



void MONTAJE()
{
    //* Declaracion de variables
    
    int i,j,nd,el,m;
    //**** NOTA : La matriz se va construyendo por filas.
    
    //* Inicializa el vector de cargas y la matriz de coeficientes
    for( i=1; i<=ngl; i++)      
    {
        b[i-1]=0.0;
        for( j=1; j<=ngl; j++)            
        {
            a[i-1][j-1]=0.0;
        }
    }
    //* Abre ficheros de coeficientes elásticos del cuerpo A
    
    //* Monta la parte correspondiente al cuerpo A del problema elastico
    printf("  Monta la parte correspondiente al cuerpo A del problema elastico\n");
    for( nd=1; nd<=nelA; nd++)      
    {
        for( el=1; el<=nelA; el++)            
        {
            //* Lee los coeficientes
            
            int arTemp_0;int arTemp_1;int arTemp_2;int arTemp_3;arTemp_2 = el*3 -2-1; for(arTemp_0=1;arTemp_0<=3;arTemp_0+=1)
            {
                arTemp_2 += 1;arTemp_3 = nd*3 -2-1; for(arTemp_1=1;arTemp_1<=3;arTemp_1+=1)
                {
                    arTemp_3 += 1;AE[arTemp_0-1][arTemp_1-1] =  AE_A[arTemp_2-1][arTemp_3-1];
                }
            }
            ;
            int arTemp_4;int arTemp_5;int arTemp_6;int arTemp_7;arTemp_6 = el*3 -2-1; for(arTemp_4=1;arTemp_4<=3;arTemp_4+=1)
            {
                arTemp_6 += 1;arTemp_7 = nd*3 -2-1; for(arTemp_5=1;arTemp_5<=3;arTemp_5+=1)
                {
                    arTemp_7 += 1;BE[arTemp_4-1][arTemp_5-1] =  BE_A[arTemp_6-1][arTemp_7-1];
                }
            }
            ;
            
            //* Mira si el nodo el pertenace a la zona actual de contacto
            if(codA[el-1][1-1] < 9)                  
            {
                //* No pertenece
                cod=codA[el-1][1-1];
                nel=0.0;
                for( m=1; m<=6; m++)                        
                {
                    cc[m-1]=ccA[el-1][m-1];
                }
                NOCONTA(&el,&nd);if(enExcepcion==1)return;
                
                //* Si pertenece
            }
            else if(codA[el-1][1-1] == 9)                  
            {
                //* Deslizamiento
                for( i=1; i<=3; i++)                        
                {
                    a[3*(nd-1)+i-1][3*(nelA+el)-2-1]=-BE[i-1][1-1]*Emedia;
                    for( j=1; j<=3; j++)                              
                    {
                        a[3*(nd-1)+i-1][3*(el-1)+j-1]=AE[i-1][j-1];
                    }
                }
            }
        }
    }
    
    //* Suma al termino independiente los coeficientes del problema termoelastico
    if(tpproTE == 1)      
    {
        for( nd=1; nd<=nelA; nd++)            
        {
            for( el=1; el<=nelA; el++)                  
            {
                int arTemp_8;int arTemp_9;int arTemp_10;arTemp_8 = 1-1; for(arTemp_9=el*3 -2;arTemp_9<=el*3;arTemp_9+=1)
                {
                    arTemp_8 += 1;for(arTemp_10=nd;arTemp_10<=nd;arTemp_10+=1)
                    {
                        CTE[arTemp_8-1] =  CTE_A[arTemp_9-1][arTemp_10-1];
                    }
                }
                ;
                int arTemp_11;int arTemp_12;int arTemp_13;arTemp_11 = 1-1; for(arTemp_12=el*3 -2;arTemp_12<=el*3;arTemp_12+=1)
                {
                    arTemp_11 += 1;for(arTemp_13=nd;arTemp_13<=nd;arTemp_13+=1)
                    {
                        DTTE[arTemp_11-1] =  DTE_A[arTemp_12-1][arTemp_13-1];
                    }
                }
                ;
                
                for( i=1; i<=3; i++)                        
                {
                    b[3*(nd-1)+i-1]=b[3*(nd-1)+i-1]+DTTE[i-1]*(tempA[el-1]-tref)-
                    CTE[i-1]*flujA[el-1];;
                }
            }
        }
    }
    //* Abre ficheros de coeficientes elasticos del cuerpo B
    //* Monta la parte correspondiente al cuerpo B del problema elastico
    printf("  Monta la parte correspondiente al cuerpo B del problema elastico\n");
    for( nd=1; nd<=nelB; nd++)      
    {
        for( el=1; el<=nelB; el++)            
        {
            //* Lee los coeficientes
            int arTemp_14;int arTemp_15;int arTemp_16;int arTemp_17;arTemp_16 = el*3 -2-1; for(arTemp_14=1;arTemp_14<=3;arTemp_14+=1)
            {
                arTemp_16 += 1;arTemp_17 = nd*3 -2-1; for(arTemp_15=1;arTemp_15<=3;arTemp_15+=1)
                {
                    arTemp_17 += 1;AE[arTemp_14-1][arTemp_15-1] =  AE_B[arTemp_16-1][arTemp_17-1];
                }
            }
            ;
            int arTemp_18;int arTemp_19;int arTemp_20;int arTemp_21;arTemp_20 = el*3 -2-1; for(arTemp_18=1;arTemp_18<=3;arTemp_18+=1)
            {
                arTemp_20 += 1;arTemp_21 = nd*3 -2-1; for(arTemp_19=1;arTemp_19<=3;arTemp_19+=1)
                {
                    arTemp_21 += 1;BE[arTemp_18-1][arTemp_19-1] =  BE_B[arTemp_20-1][arTemp_21-1];
                }
            }
            ;
            
            //* Mira si el nodo el pertenace a la zona actual de contacto
            if(codB[el-1][1-1] < 9)                  
            {
                //* No pertenece
                nel=nelA;
                cod=codB[el-1][1-1];
                for( m=1; m<=6; m++)                        
                {
                    cc[m-1]=ccB[el-1][m-1];
                }
                NOCONTA(&el,&nd);if(enExcepcion==1)return;
                //* Si pertenece
            }
            else if(codB[el-1][1-1] == 9)                  
            {
                //* Deslizamiento
                for( i=1; i<=3; i++)                        
                {
                    for( j=2; j<=3; j++)                              
                    {
                        a[3*(nelA+nd-1)+i-1][3*(nelA+el-1)+j-1]=AE[i-1][j-1];
                    }
                    a[3*(nelA+nd-1)+i-1][3*(nelA+el)-2-1]=-BE[i-1][1-1]*Emedia;
                    a[3*(nelA+nd-1)+i-1][3*el-2-1]=-AE[i-1][1-1];
                    b[3*(nelA+nd-1)+i-1]=b[3*(nelA+nd-1)+i-1]-gap[el-1]*AE[i-1][1-1];
                }
            }
        }
    }
    
    //* Suma al termino independiente los coeficientes del problema termoelastico
    if(tpproTE == 1)      
    {
        for( nd=1; nd<=nelB; nd++)            
        {
            for( el=1; el<=nelB; el++)                  
            {
                int arTemp_22;int arTemp_23;int arTemp_24;arTemp_22 = 1-1; for(arTemp_23=el*3 -2;arTemp_23<=el*3;arTemp_23+=1)
                {
                    arTemp_22 += 1;for(arTemp_24=nd;arTemp_24<=nd;arTemp_24+=1)
                    {
                        CTE[arTemp_22-1] =  CTE_B[arTemp_23-1][arTemp_24-1];
                    }
                }
                ;
                int arTemp_25;int arTemp_26;int arTemp_27;arTemp_25 = 1-1; for(arTemp_26=el*3 -2;arTemp_26<=el*3;arTemp_26+=1)
                {
                    arTemp_25 += 1;for(arTemp_27=nd;arTemp_27<=nd;arTemp_27+=1)
                    {
                        DTTE[arTemp_25-1] =  DTE_B[arTemp_26-1][arTemp_27-1];
                    }
                }
                ;
                
                for( i=1; i<=3; i++)                        
                {
                    b[3*(nelA+nd-1)+i-1]=b[3*(nelA+nd-1)+i-1]+DTTE[i-1]*(tempB[el-1]-tref)-
                    CTE[i-1]*flujB[el-1];;
                }
            }
        }
    }
    printf("fin montaje elas\n");
    
    return;
}


void INTERPR()
{
    //* Declaracion de variables
    
    int j,k;    //auxiliar
    int ix; //parametro para identificar la numeración de las variables
    //* Interpreta resultados del cuerpo A
    for( j=1; j<=nelA; j++)      
    {
        //* Mira si el nodo j perteneca a la zona de contacto actual
        if(codA[j-1][1-1] < 9)            
        {
            //* No pertenece
            ix=0.0;
            INTLIBR(&j,&ix);if(enExcepcion==1)return;
            //* Si pertenece
        }
        else if(codA[j-1][1-1] == 9)            
        {
            //* Deslizamiento
            for( k=1; k<=3; k++)                  
            {
                ccA[j-1][k-1]=x[3*(j-1)+k-1];
            }
            ccA[j-1][4-1]=x[3*(j+nelA)-2-1]*Emedia;
            ccA[j-1][5-1]=0.;
            ccA[j-1][6-1]=0.;
        }
        else            
        {
            //* Adhesion
            for( k=1; k<=3; k++)                  
            {
                ccA[j-1][k-1]=x[3*(j-1)+k-1];
                ccA[j-1][k+3-1]=x[3*(j+nelA-1)+k-1]*Emedia;
            }
        }
    }
    //* Interpreta resultados del cuerpo B
    for( j=1; j<=nelB; j++)      
    {
        //* Mira si el nodo j perteneca a la zona de contacto actual
        if(codB[j-1][1-1] < 9)            
        {
            //* No pertenece
            ix=3*nelA;
            INTLIBR(&j,&ix);if(enExcepcion==1)return;
            //* Si pertenece
        }
        else if(codB[j-1][1-1] == 9)            
        {
            //* Deslizamiento
            ccB[j-1][1-1]=gap[j-1]-ccA[j-1][1-1];
            ccB[j-1][4-1]=ccA[j-1][4-1];
            ccB[j-1][5-1]=-ccA[j-1][5-1];
            ccB[j-1][6-1]=ccA[j-1][6-1];
            for( k=2; k<=3; k++)                  
            {
                ccB[j-1][k-1]=x[3*(nelA+j-1)+k-1];
            }
        }
        else            
        {
            //* Adhesion
            ccB[j-1][1-1]=gap[j-1]-ccA[j-1][1-1];
            ccB[j-1][2-1]=ccA[j-1][2-1];
            ccB[j-1][3-1]=-ccA[j-1][3-1];
            ccB[j-1][4-1]=ccA[j-1][4-1];
            ccB[j-1][5-1]=-ccA[j-1][5-1];
            ccB[j-1][6-1]=ccA[j-1][6-1];
        }
    }
    return;
}


void DESLIZA()
{
    //* Declaracion de variables
    double tang;
    int i;  //auxiliar
    for( i=1; i<=nelpc; i++)      
    {
        if(codA[i-1][1-1] > 8)            
        {
            ve2[i-1]=ccA[i-1][2-1]-ccB[i-1][2-1];
            ve3[i-1]=ccA[i-1][3-1]+ccB[i-1][3-1];
            ve[i-1]=sqrt(pow(ve2[i-1],2)+pow(ve3[i-1],2));
            if(ve2[i-1] != 0. )                  
            {
                tang=ve3[i-1]/ve2[i-1];
                anguv[i-1]=atan(tang);
                if(ve2[i-1] < 0. )anguv[i-1]=anguv[i-1]+4.0*atan(1.0);              //3.141592654
            }
            else                  
            {
                anguv[i-1]=4.0*atan(1.0)/2.;
                if(ve3[i-1] < 0. )anguv[i-1]=anguv[i-1]+4.0*atan(1.0);              //3.141592654
            }
        }
        else            
        {
            ve[i-1]=0.;
            ve2[i-1]=0.;
            ve3[i-1]=0.;
            anguv[i-1]=0.;
        }
        anguv[i-1]=anguv[i-1]*180./4.0*atan(1.0);       //3.141592654
    }
    return;
}



void CHEQUEO()
{
    //* Declaracion de variables
    //* Inicializa los chivatos
    ifla=0;
    //* Chequeo de tracciones
    CHETRAC();if(enExcepcion==1)return;
    printf("numero de elementos que salen de la zona de contacto:%d\n",ifla);
    //* Si es negativo, retorna
    if(ifla != 0)return;
    //* Chequeo de interpenetraciones
    CHEINTE();if(enExcepcion==1)return;
    printf("numero de elementos que entran en la zona de contacto:%d\n",ifla);
    return;
}



void COMPRESIS()
{
    //* Declaracion de variables
    
    
    int nd; //auxiliar
    double mc,sigc; //Parametros de rugosidades
    double Hmicro;  //Parametros de microdurezas
    double kc,hc,hj,hb,hg;  //conductividades y conductancias contacto
    double Y,Mgas;  //interfase
    double tempg;   //Temperatura media del gas
    //* Ponemos a cero el chivato chires
    chires=0;
    //* Calculo de la resistencia térmica
    
    for( nd=1; nd<= nelpc     ; nd++)        
    {
        if(codA[nd-1][2-1] == 3)            
        {
            //*calcula parámetros de contacto DATOS: k en W/mmºC, sig en mm, Hmicro en MPa
            kc=2*lamA*lamB/(lamA+lamB);
            Hmicro=2*HAmicro*HBmicro/(HAmicro+HBmicro);
            mc=pow((pow(mA,2)+pow(mB,2)),0.50);
            sigc=pow((pow(sigA,2)+pow(sigB,2)),0.50);
            //* Calcula resistencia térmica de contacto
            
            //Ley de Yovanovich
            hc=1.25*kc*mc/sigc*pow((fabs(ccA[nd-1][4-1])/Hmicro),0.950);
            //Ley de Pilar
            //_____________________________________________________
            //hc=1.45*kc*mc/sigc*(abs(ccA(nd,4))/Hmicro)**0.985D0
            //_____________________________________________________
            
            if(TIM == 1)                
            {
                hg=0.00;
            }
            else                
            {
                Y=1.53*sigc*pow((fabs(ccA[nd-1][4-1])/Hmicro),(-0.0970));
                if(TIM == 2)                    
                {
                    tempg=(tempA[nd-1]+tempB[nd-1])/2;
                    Mgas=M0*(tempg/50)*(0.1013000/(fabs(ccA[nd-1][4-1])));
                }
                else                    
                {
                    Mgas=0.0;
                }
                hg=kg/(Y+Mgas);
            }
            hj=hc+hg;
            if(TIM == 4)                
            {
                hb=kg/(tTIM*(1-fabs(ccA[nd-1][4-1])/ETIM));
                //hb=kg/tTIM
                RTCC[nd-1]=1/hj+1/hb;
            }
            else                
            {
                RTCC[nd-1]=1/hj;
            }
            //* Resistencia ejemplo Kishimoto
            //RTCC(nd)=0.002*exp(-0.01*(abs(ccA(nd,4))))*1e6
        }
    }
    //* Compara resistencias termicas nodos en contacto
    
    for( nd=1; nd<= nelpc ; nd++)            
    {
        if(codA[nd-1][2-1] == 3)                
        {
            if(fabs((RTC[nd-1]-RTCC[nd-1])*100/RTCC[nd-1]) > (10.0) )                  
            {
                chires=1;
                goto l3434;
            }
        }
    }
    //* Asigna valores de resistencia termica
    l3434:      for( nd=1; nd<= nelpc; nd++)    
    {
        if(codA[nd-1][2-1] == 3)            
        {
            RTC[nd-1]=RTCC[nd-1];
        }
    }
    return;
}



void SALIDA_SOLU()
{
    //* Declaracion de variables
    int nd; //auxiliar
    
    //* abre ficheros de salida
    out3s = fopen(nomArchSalElas, "w");
    
    fprintf(out3s," %s",titulo);    // Formato: 5
    fprintf(out3s," Resultados problema ELÁSTICO de contacto\n");   // Formato: 10
    fprintf(out3s," \n");   // Formato: 20
    fprintf(out3s," DESPLAZAMIENTOS Y TENSIONES\n");    // Formato: 30
    fprintf(out3s," \n");   // Formato: 20
    
    //* CUERPO A
    
    fprintf(out3s," ==================== CUERPO A ====================\n"); // Formato: 35
    fprintf(out3s,"  - Zona de contacto -\n");  // Formato: 40
    
    //*  RESULTADOS EN: Movimientos [µm], Presiones [MPa], Resistencias en [mm2 ºC/W]
    //*  Zona de contacto de A
    
    for( nd=1; nd<=nelpc; nd++)    
    {
        fprintf(out3s," %4d %12.5f %12.4f %12.4f %12.4f %12.4f %12.4f %13.6f\n",nd,ccA[nd-1][1-1]*1e3,ccA[nd-1][2-1],ccA[nd-1][3-1],-ccA[nd-1][4-1],ccA[nd-1][5-1],ccA[nd-1][6-1],RTC[nd-1]);       // Formato: 100
    }
    //*  Zona libre de A
    fprintf(out3s,"  - Zona libre -\n");    // Formato: 60
    
    for( nd=nelpc+1; nd<=nelA; nd++)    
    {
        fprintf(out3s," %4d %12.5f %12.4f %12.4f %12.4f %12.4f %12.4f \n",nd,ccA[nd-1][1-1]*1e3,ccA[nd-1][2-1],ccA[nd-1][3-1],-ccA[nd-1][4-1],ccA[nd-1][5-1],ccA[nd-1][6-1]);       // Formato: 110
    }
    fprintf(out3s," \n");   // Formato: 20
    
    //* CUERPO B
    fprintf(out3s," ==================== CUERPO B ====================\n"); // Formato: 55
    fprintf(out3s,"  - Zona de contacto -\n");  // Formato: 40
    
    
    //*  Zona de contacto de B
    
    for( nd=1; nd<=nelpc; nd++)        
    {
        fprintf(out3s," %4d %12.5f %12.4f %12.4f %12.4f %12.4f %12.4f %13.6f\n",nd,ccB[nd-1][1-1]*1e3,ccB[nd-1][2-1],ccB[nd-1][3-1],-ccB[nd-1][4-1],ccB[nd-1][5-1],ccB[nd-1][6-1],RTC[nd-1]);       // Formato: 100
    }
    //*  Zona libre de B
    fprintf(out3s,"  - Zona libre -\n");    // Formato: 60
    //write(out3s,56)
    for( nd=nelpc+1; nd<=nelB; nd++)        
    {
        fprintf(out3s," %4d %12.5f %12.4f %12.4f %12.4f %12.4f %12.4f \n",nd,ccB[nd-1][1-1]*1e3,ccB[nd-1][2-1],ccB[nd-1][3-1],-ccB[nd-1][4-1],ccB[nd-1][5-1],ccB[nd-1][6-1]);       // Formato: 110
    }
    //*  Cierra fichero
    
    fclose(out3s);
    
    //* Formatos
    return;
}