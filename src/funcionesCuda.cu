#include "hip/hip_runtime.h"
#include "funcionesCuda.h"

void inicializar(EntradaCuerpo* entradaCuerpo, CONTEXTO contexto)
{
    entradaCuerpo->contexto = contexto;

    entradaCuerpo->exT  = (double **)generarMatriz(sizeof(double), 5000, 3, contexto);
    entradaCuerpo->conT = (int **)generarMatriz(sizeof(int), 5000, 3, contexto);
    entradaCuerpo->ndT  = (double **)generarMatriz(sizeof(double), 5000, 3, contexto);
    entradaCuerpo->locT = (double **)generarMatriz(sizeof(double), 5000, 9, contexto);

    entradaCuerpo->AE   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT * 3, contexto);
    entradaCuerpo->BE   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT * 3, contexto);
    entradaCuerpo->AT   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT, entradaCuerpo->nelT, contexto);
    entradaCuerpo->BT   = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT, entradaCuerpo->nelT, contexto);
    entradaCuerpo->CTE  = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT, contexto);
    entradaCuerpo->DTE  = (double **)generarMatriz(sizeof(double), entradaCuerpo->nelT * 3, entradaCuerpo->nelT, contexto);
}

void** generarMatriz(int tamanioElemento, int filas, int columnas, CONTEXTO contexto)
{
    void** variable;

    switch (contexto)
    {
        case HOST:
            variable = (void **)alloc2DOnHost(filas, columnas, tamanioElemento);
            break;

        case CUDA:
            variable = (void **)alloc2DOnDevice(filas, columnas, tamanioElemento);
            break;
    }

    return variable;
}

void** alloc2DOnHost(int rows, int columns, int sizeOfElement)
{
    int header = rows * sizeof(void*);
    int body = rows * columns * sizeOfElement;

    void** rowptr = (void**)malloc(header + body);
    char* values = (char*)(rowptr + rows);

    for (int i=0; i< rows*columns * sizeOfElement; i++)
        values[i] = 0;

    for (int i=0; i<rows; i++)
        rowptr[i] = (void*) &values[i*columns*sizeOfElement];

    return rowptr;
}

void ** alloc2DOnDevice(int rows, int cols, int sizeOfElement)
{
    int header = rows * sizeof(void*);
    int body = rows * cols * sizeOfElement;

    void** rowptr;

    hipMalloc(&rowptr, header+body);

    __organize2DMatrix<<<1,1>>>(rowptr, rows, cols, sizeOfElement);

    hipDeviceSynchronize();

    return rowptr;
}

__global__ void __organize2DMatrix(void** matrix, int rows, int cols, int sizeOfElement)
{
    char* values = (char*)(matrix + rows);

    for (int i=0; i<rows; i++)
        matrix[i] = (void*) &values[i*cols*sizeOfElement];
}

void copiarMatriz(void** matrizDestino, void** matrizOrigen, int tamanioElemento, int filas, int columnas,
                  int direccion)
{
    switch (direccion)
    {
        case MEMCPY_HOST_TO_HOST:
            memcpy(matrizDestino+filas, matrizOrigen+filas,
                   filas*columnas*tamanioElemento);
            break;
        case hipMemcpyDeviceToHost:
            hipMemcpy(matrizDestino+filas, matrizOrigen+filas,
                       filas*columnas*tamanioElemento, hipMemcpyDeviceToHost);
            break;
        case hipMemcpyHostToDevice:
            hipMemcpy(matrizDestino+filas, matrizOrigen+filas,
                       filas*columnas*tamanioElemento, hipMemcpyHostToDevice);
            break;
    }
}

void liberarMatriz(void** matriz, CONTEXTO contexto)
{
    switch (contexto)
    {
        case HOST:
            free(matriz);
            break;

        case CUDA:
            hipFree(matriz);
            break;
    }
}

void finalizar(EntradaCuerpo* entradaCuerpo)
{
    liberarMatriz((void**)entradaCuerpo->exT, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->conT, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->ndT, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->locT, entradaCuerpo->contexto);

    liberarMatriz((void**)entradaCuerpo->AE, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->BE, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->AT, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->BT, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->CTE, entradaCuerpo->contexto);
    liberarMatriz((void**)entradaCuerpo->DTE, entradaCuerpo->contexto);
}
